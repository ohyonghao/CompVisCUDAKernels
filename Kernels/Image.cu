#include "hip/hip_runtime.h"
/*
 * simple_cuda.cu
 *
 *  Created on: Nov 3, 2020
 *      Author: dpost
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <algorithm>

#include "Image.h"
using namespace std;
namespace Kernels{

//*****************************************************************************
// Global access to our mask
//*****************************************************************************

constexpr int MASK_WIDTH = 5;
constexpr int MASK_SCALE = 1 << ( ( (MASK_WIDTH - 1)  * 2 ) );

constexpr unsigned int BLOCKW = 32;
constexpr unsigned int BLOCKH = 32;
constexpr unsigned int CHANNEL = 3;
constexpr unsigned int TILEW = ((BLOCKW - (MASK_WIDTH - 1 ) * CHANNEL)/CHANNEL) ; // Number of pixels // should be 6
constexpr unsigned int TILEH = BLOCKH - MASK_WIDTH + 1;
constexpr unsigned int WORD_SIZE = 32;

//*****************************************************************************
// Functors
//*****************************************************************************

struct convert: public thrust::unary_function<float, float>
{
__host__ __device__
float operator()(float in){ return in/255.0f;}
};

struct revert: public thrust::unary_function<float, float>
{
__host__ __device__
float operator()(float in){ return in*255.0f;}
};

// Source: https://qiita.com/naoyuki_ichimura/items/8c80e67a10d99c2fb53c
inline unsigned int iDivUp( const unsigned int &a, const unsigned int &b ) { return ( a%b != 0 ) ? (a/b+1):(a/b); }

__host__
Image::Image(const Bitmap &bitmap){
    importImage(bitmap);
}

__host__
void Image::importImage(const Bitmap &bitmap){
    // TODO: Calculate importing padding known as pitch=(width + padding)
    prop.channels = bitmap.bpp();
    prop.width = bitmap.width();
    prop.height = bitmap.height();

    // Calculate the pitch
    prop.pitch = iDivUp(prop.width * prop.channels, WORD_SIZE) * WORD_SIZE;
    p_size = prop.pitch * prop.height;

    cout << "channels: " << prop.channels << endl;
    // TODO: If 4 channels, then convert to RGB
    cout << "prop.pitch: " << prop.pitch << endl;
    cout << "p_size: " << p_size << endl;

    d_image.resize(p_size);

    auto bits_in  = begin(bitmap.getBits());
    auto bits_out = begin(d_image);

    size_t i = 0;
    try{
    for( ; i < prop.height; ++i ){
        thrust::copy_n(bits_in, prop.width * prop.channels, bits_out);
        bits_in  += prop.width * prop.channels;
        bits_out += prop.pitch;
    }
    }catch(...){
        cout << "Error caught transferring host->device on " << i << "th iteration" << endl;
        auto cudaerr = hipGetLastError();
        cout << "CudaError: " << cudaerr << endl;
        throw;
    }

    cout << "d_image.size(): " << d_image.size() << endl;

    thrust::transform(thrust::device, d_image.begin(), d_image.end(), d_image.begin(), convert() );

    d_result.resize(d_image.size());
}

__host__
void Image::exportImage(Bitmap &bitmap){
    cout << "Begin Exporting Image" << endl;
    thrust::host_vector<float> h_image(d_result.begin(), d_result.end());

    auto bits_in = begin(h_image);
    auto bits_out = begin(bitmap.getBits());

//    for( size_t i = 0; i < prop.height; ++i ){
//        std::transform(h_image.begin(), h_image.end(), begin(bitmap.getBits()), revert());
//        bits_in  += prop.pitch;
//        bits_out += prop.width * prop.channels;
//    }
    for( auto i = 0; i < prop.height; ++i){
        std::transform( bits_in, bits_in + prop.width * prop.channels, bits_out, revert() );
        bits_in  += prop.pitch;
        bits_out += prop.width * prop.channels;
    }
}

__constant__ int cd_Mask[MASK_WIDTH][MASK_WIDTH];
// Width includes channels in it
__global__
void kBlur(float *d_image, float *d_result, int width, int height, int maskWidth, int pitch){

    // Threads id
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;


    // Threads output coordinates
    const int row_o = blockIdx.y * TILEH + ty;
    const int col_o = blockIdx.x * TILEW * CHANNEL + tx;

    // Threads input coordinate
    const int row_i = row_o - (MASK_WIDTH/2);
    const int col_i = col_o - CHANNEL * (MASK_WIDTH/2);

    // grab all our pixels we need
    __shared__ float s_tile[BLOCKW][BLOCKH];

    if(( row_i >= 0 ) && ( row_i < height ) &&
       ( col_i >= 0 ) && ( col_i < width ) ){
        // TODO: Replace width with pitch
        s_tile[ty][tx] = d_image[row_i * pitch + col_i];
    } else{
        // If ghost cell set to 0.0f
        s_tile[ty][tx]= 0.0f;
    }

    __syncthreads();

    float output = 0.0f;

    // Only take coordinates used in the tile, not the whole block
    // Here we are going to be careful to sum
    // only the same color channel, so move by CHANNEL
    // across the X
    if( ty < TILEH && tx < TILEW * CHANNEL ){
        for( int i = 0; i < MASK_WIDTH; ++i ){
            for( int j = 0; j < MASK_WIDTH; ++j ){
                output += cd_Mask[i][j] * s_tile[ty + i][tx + j * CHANNEL];
            }
        }
        // Do not use pitch on output
        if( row_o < height && col_o < width){
            d_result[row_o * pitch + col_o] = output / MASK_SCALE > 1.0f ? 1.0f : output / MASK_SCALE;
        }
    }
}

__host__
void CUDABlur(Bitmap &bitmap, size_t iterations){
    Image image{bitmap};
    std::vector<int> mask(MASK_WIDTH*MASK_WIDTH);
    GaussMask(mask);
    // Copy the mask to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(cd_Mask), mask.data(), MASK_WIDTH*MASK_WIDTH * sizeof(int));

    cout << "Calling CUDABlur" << endl;
    cout.flush();

    // launch kernel
    dim3 grid{iDivUp( image.width(), TILEW) * CHANNEL, iDivUp( image.height() , TILEH)};
    dim3 threadBlock{BLOCKW, BLOCKH};


    cout << "GRID_DIM: <" << grid.x << ", " << grid.y << ", " << grid.z << ">" << endl;
    cout << "BLOCK_DIM: <" << threadBlock.x << ", " << threadBlock.y << ", " << threadBlock.z << ">" << endl;
    for( auto i = 0; i < iterations; ++i ){
        kBlur<<<grid, threadBlock >>>(image.data(),
                                      image.result(),
                                      image.width() * CHANNEL,
                                      image.height(),
                                      MASK_WIDTH,
                                      image.pitch());

        hipDeviceSynchronize();
        image.swap_work();
    }
    image.swap_work();
    image.exportImage(bitmap);
}


}
